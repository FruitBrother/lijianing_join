#include<stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <string.h>
#include "hip/hip_runtime.h"
#include ""
#include <sys/time.h>
#include "table_item.h"

int count = 0;


long getCurrentTime()    
{    
   struct timeval tv;    
   gettimeofday(&tv,NULL);    
   return tv.tv_sec * 1000 + tv.tv_usec / 1000;    
} 

extern "C" void to_gpu(struct order_item* order, struct user_item* user,struct join_acce_result *result,int o_size,int u_size)
{
	
	int sum_size=o_size*u_size;
	int i,j,wh,k;
	struct result_item* g_result = (struct result_item*)malloc(sizeof(struct result_item)*sum_size);
	
	memset(g_result,0,sizeof(struct result_item)*sum_size);


	
	for(i=0;i<o_size;i++)
		for(j=0;j<u_size;j++)
		{
			wh = 0;
			
			for(k=0;k<10;k++)
			{
				if(order[i].col1[k]=='\0'&&user[j].col1[k]=='\0')
					break;
				if(order[i].col1[k]!=user[j].col1[k])
				{
					wh = 1;
					break;
				}
			}
			if(wh==0)
			{	
				for(k=0;k<10;k++)
				{
					if(order[i].col1[k]!='\0')
						{
							//g_result[count].col1[k]=order[i].col1[k];
						}
				}
				for(k=0;k<10;k++)
				{
					if(user[j].col1[k]!='\0')
						{
							//g_result[count].col2[k]=user[j].col1[k];
						}
				}
				for(k=0;k<10;k++)
				{
					if(order[i].col2[k]!='\0')
						{
							//g_result[count].col3[k]=order[i].col2[k];
						}
				}
				for(k=0;k<10;k++)
				{
					if(user[j].col2[k]!='\0')
						{
							//g_result[count].col4[k]=user[j].col2[k];
						}
				}	
				//g_result[count].flag=1;
				count++;
			}
	
		}

	result->result_len=100;

	
	result->result_addr = g_result;

	

	
	



	

	

	return ;
}
