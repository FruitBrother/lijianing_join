#include "hip/hip_runtime.h"
#include<stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <string.h>
#include "hip/hip_runtime.h"
#include ""
#include <sys/time.h>
#include "table_item.h"




long getCurrentTime()    
{    
   struct timeval tv;    
   gettimeofday(&tv,NULL);    
   return tv.tv_sec * 1000 + tv.tv_usec / 1000;       
} 
__device__ int hash(char str[])
{
	int result=0;
	int x;
	for(x=0;x<10;x++)
		{
			if(str[x]!='\0')
				result=result*10+(str[x]-'0');
			else
	
				break;		
			
		}
	return result;
}
__global__ void join(struct order_item* g_order, struct user_item* g_user,struct result_item* g_result,int g_o_size,int g_u_size,int *cnt)
{

	
	
	
	

	
        int wh = 0;
	int i,k;
	
	//int row  = blockIdx.y * blockDim.y + threadIdx.y;    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int count = 0;
	int base=((g_u_size/DIM)+1)*10*tid;
	for(i=tid;i<g_u_size;i=i+DIM)
	{
		
	
		int index=(hash(g_user[i].col1)%(g_o_size/10))*10;
		int point=index;
		//if(g_order[point].flag==0)
		//	continue;
		//else
		//{
			do
			{
				if(g_order[point].flag==0)
					break;
				wh = 0;
			
				for(k=0;k<10;k++)
				{
					if(g_order[point].col1[k]!=g_user[i].col1[k])
					{
						wh = 1;
						break;
					}
				}
				if(wh==0)
				{	
					for(k=0;k<10;k++)
					{
				
								g_result[base+count].col1[k]=g_order[point].col1[k];
					
					}
					for(k=0;k<10;k++)
					{
			
								g_result[base+count].col2[k]=g_user[i].col1[k];
		
					}
					for(k=0;k<10;k++)
					{
		
								g_result[base+count].col3[k]=g_order[point].col2[k];
			
					}
					for(k=0;k<10;k++)
					{
	
								g_result[base+count].col4[k]=g_user[i].col2[k];
						
					}	
					g_result[base+count].flag=1;
					
					count++;
					

				}
				point++;
			
			}
			while(point<(index+10));
		//}
	}
	cnt[tid]=count;
			
			
       			
}
extern "C" void to_gpu(struct order_item* order, struct user_item* user,struct join_acce_result *result,int o_size,int u_size)
{
	
	int sum_size=((u_size/DIM)+1)*DIM*10;
	int i;
	int *cnt=(int*)malloc(sizeof(int)*DIM);
	int *gpu_cnt;
	hipMalloc(&gpu_cnt, sizeof(int)*DIM);
	hipMemset(gpu_cnt,0,sizeof(int)*DIM);
	struct result_item* g_result;
	
	hipMalloc(&g_result,sizeof(struct result_item)*sum_size);
	hipMemset(g_result,0,sizeof(struct result_item)*sum_size);	
		
	struct order_item* g_order;
	struct user_item* g_user;
	
	
	
	
	hipMalloc(&g_order,sizeof(struct order_item)*o_size);

	hipMalloc(&g_user,sizeof(struct user_item)*u_size);

	hipMemcpy(g_order, order, sizeof(struct order_item)*o_size, hipMemcpyHostToDevice);
	
	hipMemcpy(g_user, user, sizeof(struct user_item)*u_size, hipMemcpyHostToDevice);
	
	//dim3 dimBlock(8,8);
	//dim3 dimGrid(1,1);
	join<<<1,8>>>(g_order,g_user,g_result,o_size,u_size,gpu_cnt);

hipMemcpy(cnt, gpu_cnt, sizeof(int)*DIM, hipMemcpyDeviceToHost);

	result->result_len=0;
	for(i=0;i<DIM;i++)
		
		{
			
			result->result_len =  result->result_len+cnt[i];
                  	
                }

	
	struct result_item* p_result = (struct result_item*)malloc(sizeof(struct result_item)*sum_size);
	hipMemcpy(p_result, g_result, sizeof(struct result_item)*sum_size, hipMemcpyDeviceToHost);
	result->result_addr = p_result;

		

	
	



	
	hipFree(g_result);
	hipFree(g_order);
	hipFree(g_user);

	return ;
}
